#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <THC/THC.h>
#include <THC/THCAtomics.cuh>
#include <THC/THCDeviceUtils.cuh>


// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename T>
__global__ void RoIPoolFForward(const int nthreads, const T* bottom_data,
    const T spatial_scale, const int channels,
    const int width, const int pooled_width,
    const T* bottom_rois, T* top_data, int* argmax_data) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int c = (index / pooled_width ) % channels;
    int n = index / pooled_width  / channels;

    const T* offset_bottom_rois = bottom_rois + n * 2;
    int roi_start_w = round(offset_bottom_rois[0] * spatial_scale);
    int roi_end_w = round(offset_bottom_rois[1] * spatial_scale);

    // Force malformed ROIs to be 1x1
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);

    T bin_size_w = static_cast<T>(roi_width)
                       / static_cast<T>(pooled_width);

    int wstart = static_cast<int>(floor(static_cast<T>(pw)
                                        * bin_size_w));

    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1)
                                     * bin_size_w));

    // Add roi offsets and clip to input boundaries
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);
    //bool is_empty = (wend <= wstart);

    // Define an empty pooling region to be zero
    //T maxval = is_empty ? 0 : -FLT_MAX;
    T maxval=0;
    // If nothing is pooled, argmax = -1 causes nothing to be backprop'd
    //int maxidx = -1;
    int maxidx = 0;
    const T* offset_bottom_data =bottom_data + c* width;

    if (wstart!=wend){
    for (int w = wstart; w < wend; ++w) {
        int bottom_index = w;
        maxval += offset_bottom_data[bottom_index];
        maxidx++;

    }
    }
    else{
        maxval +=offset_bottom_data[wstart];
        maxidx++;
    }
    top_data[index]=static_cast<T>(maxval/float(maxidx));
    argmax_data[index]=maxidx;

  }
}

template <typename T>
__global__ void RoIPoolFBackward(const int nthreads, const T* top_diff,
    const int* argmax_data, const int num_rois, const T spatial_scale,
    const int channels, const int width,
    const int pooled_width, T* bottom_diff,
    const T* bottom_rois) {
  CUDA_1D_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int c = (index / pooled_width ) % channels;
    int n = index / pooled_width  / channels;


    const T* offset_bottom_rois = bottom_rois + n * 2;
    int roi_start_w = round(offset_bottom_rois[0] * spatial_scale);
    int roi_end_w = round(offset_bottom_rois[1] * spatial_scale);
    int roi_width = max(roi_end_w - roi_start_w + 1, 1);

    T bin_size_w = static_cast<T>(roi_width)
                       / static_cast<T>(pooled_width);

    int wstart = static_cast<int>(floor(static_cast<T>(pw)
                                        * bin_size_w));

    int wend = static_cast<int>(ceil(static_cast<T>(pw + 1)
                                     * bin_size_w));
    wstart = min(max(wstart + roi_start_w, 0), width);
    wend = min(max(wend + roi_start_w, 0), width);

    int bottom_offset =  c * width;
    int top_offset    = (n * channels + c)  * pooled_width;
    const T* offset_top_diff = top_diff + top_offset;
    T* offset_bottom_diff = bottom_diff + bottom_offset;

    const int* offset_argmax_data = argmax_data + top_offset;
    int argmax = offset_argmax_data[pw];

    if (wstart!=wend){
        for (int w = wstart; w < wend; ++w) {
            int bottom_index = w;
            offset_bottom_diff[bottom_index]=static_cast<T>((offset_top_diff[pw]/float(argmax)));
        }
    }
    else{
            offset_bottom_diff[wstart]=static_cast<T>((offset_top_diff[pw]/float(argmax)));
    }

  }
}

std::tuple<at::Tensor, at::Tensor> ROIPool_forward_cuda(const at::Tensor& input,
                                const at::Tensor& rois,
                                const float spatial_scale,
                                const int pooled_width) {
  AT_ASSERTM(input.type().is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.type().is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto channels = input.size(0);
  auto width = input.size(1);

  auto output = at::empty({num_rois, channels, pooled_width}, input.options());
  auto output_size = num_rois * pooled_width * channels;
  auto argmax = at::zeros({num_rois, channels,  pooled_width}, input.options().dtype(at::kInt));

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv(output_size, 512L), 4096L));
  dim3 block(512);

  if (output.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return std::make_tuple(output, argmax);
  }

  AT_DISPATCH_FLOATING_TYPES(input.type(), "ROIPool_forward", [&] {
    RoIPoolFForward<scalar_t><<<grid, block, 0, stream>>>(
         output_size,
         input.contiguous().data<scalar_t>(),
         spatial_scale,
         channels,
         width,
         pooled_width,
         rois.contiguous().data<scalar_t>(),
         output.data<scalar_t>(),
         argmax.data<int>());
  });
  THCudaCheck(hipGetLastError());
  return std::make_tuple(output, argmax);
}

// TODO remove the dependency on input and use instead its sizes -> save memory
at::Tensor ROIPool_backward_cuda(const at::Tensor& grad,
                                 const at::Tensor& input,
                                 const at::Tensor& rois,
                                 const at::Tensor& argmax,
                                 const float spatial_scale,
                                 const int pooled_width,
                                 const int channels,
                                 const int width) {
  AT_ASSERTM(grad.type().is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(rois.type().is_cuda(), "rois must be a CUDA tensor");
  // TODO add more checks

  auto num_rois = rois.size(0);
  auto grad_input = at::zeros({channels, width}, grad.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  dim3 grid(std::min(THCCeilDiv(grad.numel(), 512L), 4096L));
  dim3 block(512);

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    THCudaCheck(hipGetLastError());
    return grad_input;
  }

  AT_DISPATCH_FLOATING_TYPES(grad.type(), "ROIPool_backward", [&] {
    RoIPoolFBackward<scalar_t><<<grid, block, 0, stream>>>(
         grad.numel(),
         grad.contiguous().data<scalar_t>(),
         argmax.data<int>(),
         num_rois,
         spatial_scale,
         channels,
         width,
         pooled_width,
         grad_input.data<scalar_t>(),
         rois.contiguous().data<scalar_t>());
  });
  THCudaCheck(hipGetLastError());
  return grad_input;
}
